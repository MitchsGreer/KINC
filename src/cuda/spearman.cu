#include "hip/hip_runtime.h"

// #include "sort.cu"






/*!
 * Compute the Spearman correlation of a cluster in a pairwise data array.
 *
 * @param x
 * @param y
 * @param labels
 * @param sampleSize
 * @param stride
 * @param cluster
 * @param minSamples
 * @param x_sorted
 * @param y_sorted
 * @param rank
 */
__device__
float Spearman_computeCluster(
   const float *x,
   const float *y,
   const char *labels,
   int sampleSize,
   int stride,
   char cluster,
   int minSamples,
   float *x_sorted,
   float *y_sorted,
   int *rank)
{
   // extract samples in pairwise cluster
   int n = 0;

   for ( int i = 0, j = 0; i < sampleSize; ++i )
   {
      if ( labels[i * stride] == cluster )
      {
         x_sorted[j] = x[i];
         y_sorted[j] = y[i];
         rank[j] = n;
         j += stride;
         n += 1;
      }
   }

   // get power of 2 size
   int N_pow2 = nextPower2(sampleSize);

   for ( int i = n * stride; i < N_pow2 * stride; i += stride )
   {
      x_sorted[i] = INFINITY;
      y_sorted[i] = INFINITY;
      rank[i] = 0;
   }

   // compute correlation only if there are enough samples
   float result = NAN;

   if ( n >= minSamples )
   {
      // execute two sorts that are the beginning of the spearman algorithm
      bitonicSortFF(N_pow2, x_sorted, y_sorted, stride);
      bitonicSortFI(N_pow2, y_sorted, rank, stride);

      // go through spearman sorted rank list and calculate difference from 1,2,3,... list
      int diff = 0;

      for ( int i = 0; i < n; ++i )
      {
         int tmp = i - rank[i * stride];
         diff += tmp*tmp;
      }

      // compute spearman coefficient
      result = 1.0 - 6.0 * diff / (n * (n*n - 1));
   }

   return result;
}






/*!
 * Compute the correlation of each cluster in a pairwise data array. The data array
 * should only contain the clean samples that were extracted from the expression
 * matrix, while the labels should contain all samples.
 *
 * @param globalWorkSize
 * @param expressions
 * @param sampleSize
 * @param in_index
 * @param clusterSize
 * @param in_labels
 * @param minSamples
 * @param out_correlations
 */
__global__
void Spearman_compute(
   int globalWorkSize,
   const float *expressions,
   int sampleSize,
   const int2 *in_index,
   char clusterSize,
   const char *in_labels,
   int minSamples,
   float *work_x,
   float *work_y,
   int *work_rank,
   float *out_correlations)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   int stride = gridDim.x * blockDim.x;

   if ( i >= globalWorkSize )
   {
      return;
   }

   // initialize workspace variables
   int N_pow2 = nextPower2(sampleSize);
   int2 index = in_index[i];
   const float *x = &expressions[index.x * sampleSize];
   const float *y = &expressions[index.y * sampleSize];
   const char *labels = &in_labels[i];
   float *x_sorted = &work_x[i];
   float *y_sorted = &work_y[i];
   int *rank = &work_rank[i];
   float *correlations = &out_correlations[i];

   for ( char k = 0; k < clusterSize; ++k )
   {
      correlations[k * stride] = Spearman_computeCluster(x, y, labels, sampleSize, stride, k, minSamples, x_sorted, y_sorted, rank);
   }
}
