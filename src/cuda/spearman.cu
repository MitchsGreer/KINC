#include "hip/hip_runtime.h"

// #include "sort.cu"






/*!
 * Compute the Spearman correlation of a cluster in a pairwise data array.
 *
 * @param x
 * @param y
 * @param labels
 * @param sampleSize
 * @param cluster
 * @param minSamples
 * @param x_sorted
 * @param y_sorted
 * @param rank
 */
__device__
float Spearman_computeCluster(
   const float *x,
   const float *y,
   const char *labels,
   int sampleSize,
   char cluster,
   int minSamples,
   float *x_sorted,
   float *y_sorted,
   int *rank)
{
   // extract samples in pairwise cluster
   int n = 0;

   for ( int i = 0; i < sampleSize; ++i )
   {
      if ( labels[i] == cluster )
      {
         x_sorted[n] = x[i];
         y_sorted[n] = y[i];
         rank[n] = n + 1;
         ++n;
      }
   }

   // get power of 2 size
   int N_pow2 = nextPower2(sampleSize);

   for ( int i = n; i < N_pow2; ++i )
   {
      x_sorted[i] = INFINITY;
      y_sorted[i] = INFINITY;
      rank[i] = 0;
   }

   // compute correlation only if there are enough samples
   float result = NAN;

   if ( n >= minSamples )
   {
      // execute two sorts that are the beginning of the spearman algorithm
      bitonicSortFF(N_pow2, x_sorted, y_sorted);
      bitonicSortFI(N_pow2, y_sorted, rank);

      // go through spearman sorted rank list and calculate difference from 1,2,3,... list
      int diff = 0;

      for ( int i = 0; i < n; ++i )
      {
         int tmp = (i + 1) - rank[i];
         diff += tmp*tmp;
      }

      // compute spearman coefficient
      result = 1.0 - 6.0 * diff / (n * (n*n - 1));
   }

   return result;
}






/*!
 * Compute the correlation of each cluster in a pairwise data array. The data array
 * should only contain the clean samples that were extracted from the expression
 * matrix, while the labels should contain all samples.
 *
 * @param globalWorkSize
 * @param expressions
 * @param sampleSize
 * @param in_index
 * @param clusterSize
 * @param in_labels
 * @param minSamples
 * @param out_correlations
 */
__global__
void Spearman_compute(
   int globalWorkSize,
   const float *expressions,
   int sampleSize,
   const int2 *in_index,
   char clusterSize,
   const char *in_labels,
   int minSamples,
   float *work_xy,
   int *work_rank,
   float *out_correlations)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;

   if ( i >= globalWorkSize )
   {
      return;
   }

   // initialize workspace variables
   int N_pow2 = nextPower2(sampleSize);
   int2 index = in_index[i];
   const float *x = &expressions[index.x * sampleSize];
   const float *y = &expressions[index.y * sampleSize];
   const char *labels = &in_labels[i * sampleSize];
   float *x_sorted = &work_xy[(2 * i + 0) * N_pow2];
   float *y_sorted = &work_xy[(2 * i + 1) * N_pow2];
   int *rank = &work_rank[i * N_pow2];
   float *correlations = &out_correlations[i * clusterSize];

   for ( char k = 0; k < clusterSize; ++k )
   {
      correlations[k] = Spearman_computeCluster(x, y, labels, sampleSize, k, minSamples, x_sorted, y_sorted, rank);
   }
}
